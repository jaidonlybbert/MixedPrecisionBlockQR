#include "hip/hip_runtime.h"
/*
Copyright (c) 2023 Jaidon Lybbert

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

/*
* CUDA implementation of the Block QR decomposition algorithm
* 
* Conventions:
*   Functions prefixed by "h_" sequentially execute on the CPU (host)
*   Functions prefixed by "dev_" execute in parallel on the GPU (device)
*/

#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <assert.h>

void h_householder_qr(float* A, float* Q, int m, int n) {
    // TASK: implement overloaded householder function, which returns Q matrix by reference
    // (rather than embedding householder vectors in lower triangular part of A)
} 

void h_householder_qr(float* A, int m, int n, int global_offset, int panel_width) {
    /*
    * Computes the QR decomposition of A using Householder reflectors.
    *
    * Reference:
    *   Golub, Van Loan. Matrix Computations, Fourth Edition. The Johns Hopkins
    *   University Press. Pg. 249. Algorithm 5.2.1
    */

    // Iterate over columns
    int r = panel_width + global_offset;
    for (int k = global_offset; k < r; k++) {
        /*
        * Compute householder vector
        */

        // Skip last transform if square matrix
        if (m == n && k == n - 1) {
            break;
        }

        // Copy the column as u - can be done in parallel
        int len = m - k;
        float* u = (float*)malloc((len) * sizeof(float));
        for (int i = 0; i < len; i++) {
            u[i] = A[n * (i + k) + k];
        }

        // Create the householder vector from the column vector
        int sign = 0;
        if (u[0] >= 0) {
            sign = 1;
        }
        else if (u[0] < 0) {
            sign = -1;
        }

        // Get the magnitude of u
        float mag = 0;
        for (int i = 0; i < len; i++) {
            mag += u[i] * u[i];
        }
        mag = sqrtf(mag);

        // Compute householder normal vector w_k
        u[0] = sign * mag + u[0]; // v overwrites u
        // Normalize
        mag = 0;
        for (int i = 0; i < len; i++) {
            mag += u[i] * u[i];
        }
        mag = sqrtf(mag);
        for (int i = 0; i < len; i++) {
            u[i] /= mag; // w_k overwrites v, here u = w_k = v_k = householder vector
        }

        /*
        * Update trailing matrix : A_k:m,k:r = A_k:m,k:r - 2V((V ^ T)(A_k:m,k:r)
        */

        // (V^T)(A_k:m,k:r) - vector matrix product
        float* temp = (float*)malloc((r - k) * sizeof(float));
        for (int col = k; col < r; col++) {
            float inner_product = 0;
            for (int row = k; row < m; row++) {
                inner_product += u[row - k] * A[row * n + col];
            }
            temp[col - k] = inner_product;
        }

        // (A_k:m,k:r) - 2 * (V)(V^T)(A_k:m,k:r)
        float* temp2 = (float*)malloc((r - k) * (m - k) * sizeof(float));
        for (int row = k; row < m; row++) {
            for (int col = k; col < r; col++) {
                temp2[(row - k) * (r - k) + (col - k)] = u[row - k] * temp[col - k];
                A[row * n + col] = A[row * n + col] - 2 * temp2[(row - k) * (r - k) + (col - k)];
            }
        }

        // Copy householder vector (vk) to lower triangular portion of A
        for (int row = k + 1; row < k + len + 1; row++) {
            A[row * n + k] = u[row - k - 1];
        }

        free(temp);
        free(temp2);
        free(u);
    }
}

void h_wy_transform(float* h_A, float** h_Q, int m, int n, int global_offset, int panel_width)
{
    float* W = (float*)malloc((m - global_offset) * panel_width * sizeof(float));
    float* Y = (float*)malloc((m - global_offset) * panel_width * sizeof(float));
    float* z = (float*)malloc((m - global_offset) * sizeof(float));
    float* W_Yt = (float*)malloc((m - global_offset) * (m - global_offset) * sizeof(float)); // temporary matrix W * Y^T

    // Y = w_1
    for (int i = 0; i < m - global_offset; i++) {
        Y[i * panel_width] = h_A[(i + global_offset + 1) * n + global_offset];
        W[i * panel_width] = 2 * h_A[(i + global_offset + 1) * n + global_offset];
    }

    // Iterate over columns of panel and update W, Y
    for (int i = 1; i < panel_width; i++) { // cols of panel
        // Calculate z = 2 * (I_m - WY^T)w_i

        // Im - WY^T (classic "triply-nested-loop")
        for (int row = 0; row < m - global_offset; row++) { // rows of W_Yt
            for (int col = 0; col < m - global_offset; col++) { // cols of W_Yt
                // compute each inner product
                float inner_product = 0;
                for (int idx = 0; idx < i; idx++) { // rows of W
                    inner_product += W[row * panel_width + idx] * Y[col * panel_width + idx];
                }
                if (row == col) { // Im is 1
                    W_Yt[row * (m - global_offset) + col] = 1 - inner_product; // Im - WY^T
                }
                else { // Im is zero
                    W_Yt[row * (m - global_offset) + col] = -inner_product;
                }
            }
        }

        // 2 * (Im - WY^T)w_i (matrix-vector product)
        for (int row = 0; row < (m - global_offset); row++) {
            float inner_product = 0;
            // start at i, because w_i is zero before then
            for (int col = i; col < (m - global_offset); col++) {
                inner_product += W_Yt[row * (m - global_offset) + col] * h_A[(global_offset + col + 1) * n + global_offset + i];
            }
            z[row] = 2 * inner_product;
        }

        // Copy z to W
        for (int idx = 0; idx < (m - global_offset); idx++) {
            if (idx < (i)) {
                Y[idx * panel_width + i] = 0;
            }
            else {
                Y[idx * panel_width + i] = h_A[(global_offset + idx + 1) * n + global_offset + i];
            }
            W[idx * panel_width + i] = z[idx];
        }
    }

    // Im - WY^T (classic "triply-nested-loop")
    for (int row = 0; row < m - global_offset; row++) { // rows of W_Yt
        for (int col = 0; col < m - global_offset; col++) { // cols of W_Yt
            // compute each inner product
            float inner_product = 0;
            for (int idx = 0; idx < panel_width; idx++) { // cols of W
                inner_product += W[row * panel_width + idx] * Y[col * panel_width + idx];
            }
            if (row == col) { // Im is 1
                W_Yt[row * (m - global_offset) + col] = 1 - inner_product; // Im - WY^T
            }
            else { // Im is zero
                W_Yt[row * (m - global_offset) + col] = -inner_product;
            }
        }
    }

    free(W);
    free(Y);
    free(z);
    //free(W_Yt);
    *h_Q = W_Yt;
}


__global__ 
void dev_householder_qr(float *dev_A, int m, int n, int global_offset) {
    /*
    * Computes the QR decomposition of A using Householder reflectors.
    * 
    * Reference: 
    *   Golub, Van Loan. Matrix Computations, Fourth Edition. The Johns Hopkins 
    *   University Press. Pg. 249. Algorithm 5.2.1
    */

    // Iterate over columns
    for (int k = global_offset; k < n; k++) {
        /*
        * Compute householder vector
        */

        // Skip last transform is square matrix
        if (m == n && k == n - 1) {
            break;
        }

        // Copy the column as u - can be done in parallel
        int len = m - k;
        float* u = (float*)malloc((len) * sizeof(float));
        for (int i = 0; i < len; i++) {
            u[i] = dev_A[n * (i+k) + k];
        }

        // Create the householder vector from the column vector
        int sign = 0;
        if (u[0] >= 0) {
            sign = 1;
        }
        else if (u[0] < 0) {
            sign = -1;
        }

        // Get the magnitude of u
        float mag = 0;
        for (int i = 0; i < len; i++) {
            mag+=u[i] * u[i];
        }
        mag = sqrtf(mag);

        // Compute householder normal vector w_k
        u[0] = sign * mag + u[0]; // v overwrites u
        // Normalize
        mag = 0;
        for (int i = 0; i < len; i++) { // TASK: implement parallel algorithm in CUDA to replace for loop
            mag += u[i] * u[i];
        }
        mag = sqrtf(mag);
        for (int i = 0; i < len; i++) { // TASK: implement parallel algorithm in CUDA to replace for loop
            u[i] /= mag; // w_k overwrites v, here u = w_k = v_k = householder vector
        }

        /*
        * Update trailing matrix : A_k:m, k : n = A_k:m,k:n - 2V((V ^ T)(A_k:m, k : n)
        */

        // (V^T)(A_k:m,k:n) - vector matrix product
        float* temp = (float*)malloc((n - k) * sizeof(float));
        for (int col = k; col < n; col++) { // TASK: implement parallel algorithm in CUDA to replace for loop
            float inner_product = 0;
            for (int row = k; row < m; row++) {
                inner_product += u[row - k] * dev_A[row * n + col];
            }
            temp[col-k] = inner_product;
        }
        
        // (A_k:m,k:n) - 2 * (V)(V^T)(A_k:m,k:n)
        float* temp2 = (float*)malloc((n - k) * (m - k) * sizeof(float));
        for (int row = k; row < m; row++) { // TASK: implement parallel algorithm in CUDA to replace for loop
            for (int col = k; col < n; col++) {
                temp2[(row - k) * (n - k) + (col - k)] = u[row-k] * temp[col-k];
                dev_A[row * n + col] = dev_A[row * n + col] - 2 * temp2[(row - k) * (n - k) + (col - k)];
            }
        }

        // Copy householder vector (vk) to lower triangular portion of A
        for (int row = k + 1; row < k + len + 1; row++) { // TASK: implement parallel algorithm in CUDA to replace for loop
            dev_A[row * n + k] = u[row - k - 1];
        }

        free(temp);
        free(temp2);
        free(u);
    }
}

float* h_generate_random_matrix(int height, int width) {
    /*
    * Returns pointer to random float matrix of dimensions HeightxWidth
    */

    float* matrix = (float*)malloc(height * width * sizeof(float));

    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            matrix[row * width + col] = 0; // TASK: randomize this number
        }
    }

    return matrix;
}

void read_euroc_jacobian(const char filename[], int* rows, int* cols, double** matrix) {
    /*
    * Reads text file containing jacobian matrices from the Euroc dataset, and returns pointer to matrix
    */

    std::ifstream fin;

    std::string line;

    fin.open(filename);

    if (!fin) {
        printf("File not found.");
    }

    assert(fin);

    // Read first line to get dimensions
    getline(fin, line);

    std::cout << line << std::endl;
    int start = line.find(" ");
    int end = line.find(" ");

    std::string rows_str = line.substr(0, start);
    std::string cols_str = line.substr(start + 1, end);

    std::cout << rows_str << std::endl;
    std::cout << cols_str << std::endl;

    *cols = std::stoi(cols_str);
    *rows = std::stoi(rows_str);

    printf("Rows: %d\nCols: %d\n", *rows, *cols);

    // Allocate memory for matrix
    *matrix = (double*)malloc((*rows) * (*cols) * sizeof(double));

    for (int row = 0; row < (*rows); row++) {
        for (int col = 0; col < (*cols); col++) {
            (*matrix)[row * (*cols) + col] = (double)0.0;
        }
    }

    int linecount = 0;
    while (getline(fin, line)) {
        //std::cout << line << std::endl;

        std::wstring::size_type pos = line.find_first_not_of(' ');
        line = line.substr(pos);
        pos = line.find(' ');
        std::string row_idx_str = line.substr(0, pos);
        line = line.substr(pos);

        pos = line.find_first_not_of(' ');
        line = line.substr(pos);
        pos = line.find(' ');
        std::string col_idx_str = line.substr(0, pos);
        line = line.substr(pos);

        pos = line.find_first_not_of(' ');
        line = line.substr(pos);
        pos = line.find(' ');
        std::string val_str = line.substr(0, pos);

        //std::cout << row_idx_str << std::endl;
        //std::cout << col_idx_str << std::endl;
        //std::cout << val_str << std::endl;

        //printf("Row idx: %d\nCol idx: %d\nVal: %.15f\n", std::stoi(row_idx_str), std::stoi(col_idx_str), std::stod(val_str));

        int row_idx = std::stoi(row_idx_str);
        int col_idx = std::stoi(col_idx_str);
        double val = std::stod(val_str);

        (*matrix)[row_idx * (*cols) + col_idx] = val;
        linecount++;
    }

    printf("Total linecount: %d\n", linecount);
}


__global__
void dev_apply_qt(float* dev_A, float* dev_Q, int m, int n, int tau) {
    // Collaboratively load householder vectors vk from global memory to shared memory
    // Construct W, K from householder vectors
    // Construct Q
    // Collaboratively load matrix A to shared memory
    // Perform tiled GMMULT TensorCore warp-level mixed precision fused multiply add operations to update Q and A
    // Update matrix Q, A in global memory
}


void dev_block_qr(float* dev_A, float* dev_Q, int m, int n, int r) {
    /*
    * Kernel to compute QR decomposition with Block QR algorithm
    */

    int k = 0;

    // initialize Q, lambda, k
    while (int lambda = 0 <= n) {
        // set panel offset
        int tau = (lambda + r - 1 < n) ? (lambda + r - 1) : n;
        k += 1;

        dim3 GridDim(1, 1, 1);
        dim3 BlockDim(m, n, 1);

        dev_householder_qr<<<GridDim, BlockDim>>>(dev_A, m, n, lambda);

        hipDeviceSynchronize();

        // Q is stored in factored form in lower triangular portion of dev_A
        // R is stored in upper triangular portion of dev_A
        //apply_qt<<<GridDim, BlockDim>>>(dev_A, m, n, tau);

        hipDeviceSynchronize();

        // increment panel offset
    }
}

void test_dev_householder_qr() {
    int rows, cols;
    double* mtx;

    read_euroc_jacobian("C:\\Users\\jaido\\source\\MixedPrecisionBlockQR\\Cuda\\jacobians\\A_000000100.txt", &rows, &cols, &mtx);

    int m = 3;
    int n = 3;

    // Initialize test matrix A input on Host
    float h_A_in[3][3] = {
        {12, -51, 4},
        {6, 167, -68},
        {-4, 24, -41},
    };

    float* h_A_out = (float*)malloc((m+1) * n * sizeof(float)); // extra row gives room for storing householder vectors in lower triangular portion of A
    float* h_Q_out = (float*)malloc(m * m * sizeof(float));

    // Allocate device memory for input matrix
    float* dev_A;
    float* dev_Q; // Matrix Q in A=QR

    hipMalloc(&dev_Q, m * m * sizeof(float));
    hipMalloc(&dev_A, (m+1) * n * sizeof(float));

    // Copy input matrix to device Global memory
    hipMemcpy(dev_A, h_A_in, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Call kernel to collaboratively copy input matrix from Global memory to Shared memory
    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(1, 1, 1);
    // TASK: Time execution of the following kernel call
    dev_householder_qr <<<DimGrid, DimBlock >> > (dev_A, m, n, 0);

    hipDeviceSynchronize();

    hipMemcpy(h_A_out, dev_A, (m+1) * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_Q_out, dev_Q, m * m * sizeof(float), hipMemcpyDeviceToHost);

    // TASK: Compute error
    // TASK: Write results to log file

    //h_wy_transform(h_A_out, m, n, 0, n);

}

void h_identity_mtx(float* I, int m, int n) {
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            if (row == col) {
                I[row * n + col] = 1;
            }
            else {
                I[row * n + col] = 0;
            }
        }
    }
}

void h_block_qr(float* A, float* Q, int m, int n, int r) {
    /*
    * Sequential version of the block QR algorithm, runs on CPU
    */

    // initialize Q, lambda, k
    h_identity_mtx(Q, m, m);
    float* panel_Q = NULL;
    int lambda = 0;
    while (lambda < n) { // panel starts at lambda
        int tau = (lambda + r < n) ? (lambda + r) : n; // panel ends at tau

        // Q is stored in factored form in lower triangular portion of dev_A
        // R is stored in upper triangular portion of dev_A
        h_householder_qr(A, m, n, lambda, r);

        // Get panel Q from factors
        h_wy_transform(A, &panel_Q, m, n, lambda, r); // dim panel_Q: (m-lambda)x(m-lambda)

        // Update matrix A
        float* A_old = (float*)malloc(m * n * sizeof(float));
        memcpy(A_old, A, m * n * sizeof(float));
        for (int row = lambda; row < m; row++) {
            for (int col = tau; col < n; col++) {
                float inner_product = 0;
                for (int inner_dim = 0; inner_dim < (m - lambda); inner_dim++) {
                    inner_product += panel_Q[(inner_dim) * (m - lambda) + (row - lambda)] * A_old[(row + inner_dim) * n + col];
                }
                A[row * n + col] = inner_product;
            }
        }
        free(A_old);

        // Update global Q
        float* Q_old = (float*)malloc(m * m * sizeof(float)); 
        memcpy(Q_old, Q, m * m * sizeof(float));
        for (int row = 0; row < m; row++) {
            for (int col = lambda; col < m; col++) {
                float inner_product = 0;
                for (int inner_dim = 0; inner_dim < (m - lambda); inner_dim++) {
                    inner_product += Q_old[row * n + inner_dim] * panel_Q[(inner_dim * (m-lambda)) + (col-lambda)];
                }
                Q[row * m + col] = inner_product;
            }
        }
        free(Q_old);

        // increment panel offset
        lambda = tau;
    }
}

void h_mmult(float* A, float* B, float* C, int m, int n, int k) {
    /*
    * A - mxk
    * B - kxn
    * C - mxn
    * 
    * C = AB
    */

    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            float inner_product = 0;
            for (int inner_idx = 0; inner_idx < k; inner_idx++) {
                inner_product += A[row * k + inner_idx] * B[(inner_idx) * n + col];
            }
            C[row * n + col] = inner_product;
        }
    }
}

void h_matrix_subtract(float* A, float* B, float* C, int m, int n) {
    /*
    * Dimensions all match, element-wise subtraction
    * 
    * C = A - B
    */

    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            C[row * n + col] = A[row * n + col] - B[row * n + col];
        }
    }
}

float h_matrix_norm(float* A, int m, int n) {
    /*
    * A shape: mxn
    * 
    * norm = ||A||
    */

    float norm = 0;
    float squared_sum = 0;
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            squared_sum += A[row * n + col] * A[row * n + col];
        }
    }
    return sqrtf(squared_sum);
}

void h_matrix_cpy(float* A, float* B, int m, int n) {
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            B[row * n + col] = A[row * n + col];
        }
    }
}

void h_strip_R_from_A(float* A, float* R, int m, int n) {
    /*
    * Removes householder vectors from lower triangular section of A
    */

    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            if (row <= col) {
                R[row * n + col] = A[row * n + col];
            }
            else {
                R[row * n + col] = 0;
            }
        }
    }
}

float h_backward_error(float* A, float* R, float* Q, int m, int n) {
    // Computes || A - QR||/||A ||

    float* QR = (float*)malloc(m * n * sizeof(float));
    float* A_sub_QR = (float*)malloc(m * n * sizeof(float));
    
    h_mmult((float*)Q, R, QR, m, n, m);
    h_matrix_subtract((float*)A, QR, A_sub_QR, m, n);

    float backward_error = (h_matrix_norm(A_sub_QR, m, n) / h_matrix_norm((float*)A, m, n));

    free(QR);
    free(A_sub_QR);

    return backward_error;
}

float h_error_2() {
    // TASK: Compute second type of error for QR result (there are 3 types - source: paper reffered by Tong)
    return 0;
}

float h_error_3() {
    // TASK: Compute third type of error for QR result (there are 3 types - source: paper reffered by Tong)
    return 0;
}

void test_h_mmult() {
    float A[3][3] = {
        {1, 2, 3},
        {1, 2, 3},
        {1, 2, 3}
    };

    int m = 3;
    int n = 3;
    int k = 3;

    float* C = (float*)malloc(m * n * sizeof(float));

    h_mmult((float*)A, (float*)A, C, m, n, k);
}

void test_h_householder_qr() {
    /*
    * Test host version of householder QR
    */

    // TASK: iterate over many matrix sizes, & test matrices from Tong
    printf("\nTesting sequential householder QR...\n");

    float A_in[6][6] = {
        {10,20,30,40,50,60},
        {32,32,44,55,66,35},
        {23,66,74,64,45,65},
        {67,28,46,26,46,42},
        {95,95,52,88,65,11},
        {75,53,96,47,32,32},
    };

    int m = 6;
    int n = 6;
    int r = 3;
    for (int global_offset = 0; global_offset < 6; global_offset++) {
        float* Q = (float*)malloc(m * m * sizeof(float));
        float* R = (float*)malloc(m * n * sizeof(float));
        float* A_out = (float*)malloc((m + 1) * n * sizeof(float));

        h_matrix_cpy((float*)A_in, A_out, m, n);

        //h_block_qr((float*)A, Q, m, n, r);
        h_householder_qr((float*)A_out, m, n, global_offset, r);

        h_wy_transform(A_out, &Q, m, n, global_offset, r);

        h_strip_R_from_A((float*)A_out, R, m, n);

        float backward_error = h_backward_error((float*)A_in, R, Q, m, n);
        printf("Backward error: %f\n", backward_error);
        printf("Sequential householder QR finished...\n");

        // TASK: write results to log file

        free(Q);
        free(R);
        free(A_out);
    }


}


void test_h_wy_transform() {
    // Initialize test matrix A input on Host
    // TASK: iterate over many matrix sizes
    int m = 3;
    int n = 3;

    // TASK: use h_generate_random_matrix to randomize input matrix
    float h_A_in[3][3] = {
        {12, -51, 4},
        {6, 167, -68},
        {-4, 24, -41},
    };

    float* h_A_out = (float*)malloc((m + 1) * n * sizeof(float)); // extra row (m+1) gives room for storing householder vectors in lower triangular portion of A
    float* h_R = (float*)malloc(m * n * sizeof(float));
    float* h_Q_out = NULL; // pointer to Q is returned by h_wy_transform

    // TASK: how can we test wy_transform without depending on using QR function?
    h_householder_qr((float*)h_A_in, m, n, 0, n);

    h_wy_transform((float*)h_A_out, &h_Q_out, m, n, 0, n);

    h_strip_R_from_A(h_A_out, h_R, m, n);

    float backward_error = h_backward_error((float*)h_A_in, h_R, h_Q_out, m, n);

    // TASK: print matrix size & backward error

    free(h_A_out);
    free(h_Q_out);
    free(h_R);
}

void h_write_results_to_log(int height, int width, float time_ms, float flops_per_second, float backward_error) {
    // TASK: write arguments to log file
}

float h_qr_flops_per_second(float time_ms, int m, int n) {
    /*
    * Computes FLOPs / second for householder QR given matrix dimensions and execution time
    * 
    * TASK: Verify equation and provide academic reference for equation (textbook or paper)
    */
    return (4. * (pow<float>(m, 2) * n - m * pow<float>(n, 2) + pow<float>(n, 3) / 3.));
}

void test_h_block_qr() {
    /*
    * Test host version of householder QR
    */

    printf("\nTesting sequential block QR...\n");

    // TASK: use read_euroc_jacobian to load test matrices
    float A_in[6][6] = {
        {10,20,30,40,50,60},
        {32,32,44,55,66,35},
        {23,66,74,64,45,65},
        {67,28,46,26,46,42},
        {95,95,52,88,65,11},
        {75,53,96,47,32,32},
    };

    int m = 6;
    int n = 6;
    int r = 3;

    float* Q = (float*)malloc(m * m * sizeof(float));
    float* R = (float*)malloc(m * n * sizeof(float));
    float* A_out = (float*)malloc((m + 1) * n * sizeof(float));

    h_matrix_cpy((float*)A_in, A_out, m, n);

    float time_ms = 0; // TASK: Time how long the QR function takes to execute

    h_block_qr((float*)A_out, Q, m, n, r);

    float flops_per_second = h_qr_flops_per_second(time_ms, m, n); // TASK: verify equation in function through research

    h_strip_R_from_A((float*)A_out, R, m, n);

    float backward_error = h_backward_error((float*)A_in, R, Q, m, n);

    // TASK: Implement following function to write results to log file
    h_write_results_to_log(m, n, time_ms, flops_per_second, backward_error);

    printf("Sequential block QR finished...\n");
    printf("Backward error: %f", backward_error);

    free(Q);
    free(R);
    free(A_out);
}

int main() {
    test_dev_householder_qr();
    test_h_mmult();
    test_h_householder_qr();
    test_h_block_qr();
}
