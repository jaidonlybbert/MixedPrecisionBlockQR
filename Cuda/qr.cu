#include "hip/hip_runtime.h"
/*
Copyright (c) 2023 Jaidon Lybbert <jaidonlybbert@gmail.com>

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

/*
* CUDA implementation of the Block QR decomposition algorithm
* 
* Conventions:
*   Functions prefixed by "h_" sequentially execute on the CPU (host)
*   Functions prefixed by "dev_" execute in parallel on the GPU (device)
*/

// CUDA includes
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <nvtx3/nvToolsExt.h>
#include <mma.h>

#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <assert.h>
#include <cstdlib>
#include <vector>
#include <chrono>
#include <dirent.h>
#include <iomanip>

#define TC_TILE_M 16
#define TC_TILE_N 16
#define TC_TILE_K 16
#define WARP_SIZE 32

// Shared memory tile sizes for tiled global memory mmult
#define GMEM_MMULT_TILE_WIDTH 16
#define GMEM_MMULT_TILE_HEIGHT 16

// Thread block sizes for TC MMULT
// Shrink to get more shared memory per warp, expand to increase memory re-use
#define TC_MMULT_THREAD_BLOCK_WIDTH 4 * WARP_SIZE
#define TC_MMULT_THREAD_BLOCK_HEIGHT 4

// Thread block sizes for array cpy
#define CPY_ARRAY_BLOCK_WIDTH 32
#define CPY_ARRAY_BLOCK_HEIGHT 32

#define TOP_LEFT 1
#define BOTTOM_RIGHT 0

typedef void QR_FUNC(int, int, int);
typedef void MMULT_FUNC(int, int, int);

void h_write_results_to_log(int height, int width, float time_ms, float flops_per_second, float backward_error, std::string file_name = "logFile") {
    //write arguments to log file
    std::vector<double> params = { height * 1.0, width * 1.0, time_ms, flops_per_second, backward_error };
    std::string path = "log/" + file_name + ".txt";
    std::ifstream file(path);
    std::string line = "";

    if (!file.good()) {
        line += "rows,cols,runtime,flops,error\n";
    }

    for (int i = 0; i < params.size(); i++)
    {
        line += std::to_string(params[i]);
        if (i != params.size() - 1) {
            line += ',';
        }
    }
    line += "\n";


    std::ofstream logFile;
    logFile.open(path, std::ios::app);
    logFile << line;
    logFile.close();
}

void h_identity_mtx(float* I, int m, int n) {
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            if (row == col) {
                I[row * n + col] = 1;
            }
            else {
                I[row * n + col] = 0;
            }
        }
    }
}

void h_mmult(float* A, float* B, float* C, int m, int n, int k) {
    /*
    * A - mxk
    * B - kxn
    * C - mxn
    *
    * C = AB
    */

    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            float inner_product = 0;
            for (int inner_idx = 0; inner_idx < k; inner_idx++) {
                inner_product += A[row * k + inner_idx] * B[(inner_idx)*n + col];
            }
            C[row * n + col] = inner_product;
        }
    }
}

void h_mmult_transpose_A(float* A, float* B, float* C, int m) {
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < m; col++) {
            float inner_product = 0;
            for (int inner_idx = 0; inner_idx < m; inner_idx++) {
                inner_product += A[(inner_idx)* m + row] * B[(inner_idx)*m + col];
            }
            C[row * m + col] = inner_product;
        }
    }
}

void h_matrix_subtract(float* A, float* B, float* C, int m, int n) {
    /*
    * Dimensions all match, element-wise subtraction
    *
    * C = A - B
    */

    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            C[row * n + col] = A[row * n + col] - B[row * n + col];
        }
    }
}

float h_matrix_norm(float* A, int m, int n) {
    /*
    * A shape: mxn
    *
    * norm = ||A||
    */

    float norm = 0;
    float squared_sum = 0;
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            squared_sum += A[row * n + col] * A[row * n + col];
        }
    }
    return sqrtf(squared_sum);
}

void h_matrix_cpy(float* A, float* B, int m, int n) {
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            B[row * n + col] = A[row * n + col];
        }
    }
}

void h_strip_R_from_A(float* A, float* R, int m, int n) {
    /*
    * Removes householder vectors from lower triangular section of A
    */

    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            if (row <= col) {
                R[row * n + col] = A[row * n + col];
            }
            else {
                R[row * n + col] = 0;
            }
        }
    }
}

float h_qr_flops_per_second(float time_ms, int m, int n) {
    /*
    * Computes FLOPs / second for householder QR given matrix dimensions and execution time
    *
    * TASK21 2 Mike: Verify equation and provide academic reference for equation (textbook or paper)
    */
    return (4. * (pow<float>(m, 2) * n - m * pow<float>(n, 2) + pow<float>(n, 3) / 3.)) / (time_ms / 1000);
}

float h_backward_error(float* A, float* R, float* Q, int m, int n) {
    // Computes || A - QR||/||A ||

    float* QR = (float*)malloc(m * n * sizeof(float));
    float* A_sub_QR = (float*)malloc(m * n * sizeof(float));
    bool pass = false;
    const double error_limit = 1.1920928955078125e-07;
    h_mmult((float*)Q, R, QR, m, n, m);
    h_matrix_subtract((float*)A, QR, A_sub_QR, m, n);

    float a_norm = h_matrix_norm((float*)A, m, n);

    float backward_error = (h_matrix_norm(A_sub_QR, m, n) / a_norm);
    if (backward_error <= error_limit * m){
            pass = true;
    }
    printf("||A - QR||/||A|| = %e Error Criteria: %s\n", backward_error, pass ? "True" : "False");
    free(QR);
    free(A_sub_QR);

    return backward_error;
}

float h_error_2(float* Q, int m) {

    // ||Q^T @ Q - Im||
    const double error_limit = pow<double>(2, -23);//1.1920928955078125e-07;
    bool pass = false;
    float* Qt_Q = (float*)malloc(m * m * sizeof(float));
    float* Im = (float*)malloc(m * m * sizeof(float));
    float* Qt_Q_sub_Im = (float*)malloc(m * m * sizeof(float));

    h_mmult_transpose_A(Q, Q, Qt_Q, m);
    h_identity_mtx(Im, m, m);
    h_matrix_subtract(Qt_Q, Im, Qt_Q_sub_Im, m, m);

    float max_error = 0;
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < m; col++) {
            if (Qt_Q_sub_Im[row * m + col] > max_error) {
                max_error = Qt_Q_sub_Im[row * m + col];
            }
        }
    }
    if (max_error <= error_limit * m){
            pass = true;
    }
    printf("||QT @ Q - Im|| = %E Error Criteria: %s\n", max_error, pass ? "True" : "False: should be less than ");

    if (!pass) {
        printf("%.2E\n", error_limit * m);
    }

    free(Qt_Q);
    free(Im);
    free(Qt_Q_sub_Im);

    return max_error;
}

float h_error_3(float* R, int m, int n) {
    // Compute third type of error for QR result
    // ||L|| < m * 2E-23
    const double error_limit = 1.1920928955078125e-07;
    bool pass = false;
    float* L = (float*)malloc(m * n * sizeof(float));
    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            if (col < row){
                L[row * n + col] = R[row * n + col];
            }
            else{
                L[row * n + col] = 0;
            }
        }
    }
    float error3 = (h_matrix_norm(L, m, n));
    if (error3 <= error_limit * m){
	    pass = true;
    }
    printf("||L|| = %e Error Criteria: %s\n", error3, pass ? "True" : "False");
    free(L);

    return error3;
}

void h_householder_qr(float* A, int m, int n, int global_offset, int panel_width) {
    /*
    * Computes the QR decomposition of A using Householder reflectors.
    *
    * Reference:
    *   Golub, Van Loan. Matrix Computations, Fourth Edition. The Johns Hopkins
    *   University Press. Pg. 249. Algorithm 5.2.1
    */

    nvtxRangePush(__func__);

    // Iterate over columns
    int r = (panel_width + global_offset) > n ? n: panel_width + global_offset;
    for (int k = global_offset; k < r; k++) {
        /*
        * Compute householder vector
        */

        // Skip last transform if square matrix
        //if (m == n && k == n - 1) {
        //    break;
        //}

        // Copy the column as u - can be done in parallel
        int len = m - k;
        float* u = (float*)malloc((len) * sizeof(float));
        for (int i = 0; i < len; i++) {
            u[i] = A[n * (i + k) + k];
        }

        // Create the householder vector from the column vector
        int sign = 0;
        if (u[0] >= 0) {
            sign = 1;
        }
        else if (u[0] < 0) {
            sign = -1;
        }

        // Get the magnitude of u
        float mag = 0;
        for (int i = 0; i < len; i++) {
            mag += u[i] * u[i];
        }
        mag = sqrtf(mag);

        // Compute householder normal vector w_k
        u[0] = sign * mag + u[0]; // v overwrites u
        // Normalize
        mag = 0;
        for (int i = 0; i < len; i++) {
            mag += u[i] * u[i];
        }
        mag = sqrtf(mag);
        for (int i = 0; i < len; i++) {
            u[i] /= mag; // w_k overwrites v, here u = w_k = v_k = householder vector
        }

        /*
        * Update trailing matrix : A_k:m,k:r = A_k:m,k:r - 2V((V ^ T)(A_k:m,k:r)
        */

        // (V^T)(A_k:m,k:r) - vector matrix product
        float* temp = (float*)malloc((r - k) * sizeof(float));
        for (int col = k; col < r; col++) {
            float inner_product = 0;
            for (int row = k; row < m; row++) {
                inner_product += u[row - k] * A[row * n + col];
            }
            temp[col - k] = inner_product;
        }

        // (A_k:m,k:r) - 2 * (V)(V^T)(A_k:m,k:r)
        float* temp2 = (float*)malloc((r - k) * (m - k) * sizeof(float));
        for (int row = k; row < m; row++) {
            for (int col = k; col < r; col++) {
                temp2[(row - k) * (r - k) + (col - k)] = u[row - k] * temp[col - k];
                A[row * n + col] = A[row * n + col] - 2 * temp2[(row - k) * (r - k) + (col - k)];
            }
        }

        // Copy householder vector (vk) to lower triangular portion of A
        for (int row = k + 1; row < k + len + 1; row++) {
            A[row * n + k] = u[row - k - 1];
        }

        free(temp);
        free(temp2);
        free(u);
    }

    nvtxRangePop();
}


void h_q_backward_accumulation(float* h_A, float** h_Q, int m, int n) {
    /*
    * "Backward accumulation" of Q from householder vectors stored in lower trapezoidal region
    *   of A, after householder QR
    * 
    * Reference:
    *   Golub, Van Loan. Matrix Computations, Fourth Edition. The Johns Hopkins
    *   University Press. Pg. 238. Algorithm 5.1.5
    */

    // Initialize Q as identity
    *h_Q = (float*)malloc(m * m * sizeof(float));
    h_identity_mtx(*h_Q, m, m);

    // Declare temporary vectors
    float* v;
    float beta;

    for (int j = n - 1; j >= 0; j--) { // iterate over householder vectors stored in lower part of A
        int v_length = m - j; // v is the householder vector, smallest first
        v = (float*)malloc((m - j) * sizeof(float));

        // Q = (Im - 2v(v^T))Q
        // Q = Q_j:m,j:m - 2V @ ((V^T) @ Q_j:m,j:m)

        // (V^T) @ Q_j:m,j:m
        float* temp = (float*)malloc((m - j) * sizeof(float));
        for (int col = j; col < m; col++) {
            float inner_product = 0;
            for (int row = j; row < m; row++) {
                inner_product += h_A[(row + 1) * n + j] * (*h_Q)[row * m + col];
            }
            temp[col - j] = inner_product;
        }

        // Q_j:m,j:m = Q_j:m,j:m - 2 * V @ ((V^T) @ Q_j:m,j:m)
        for (int row = j; row < m; row++) {
            for (int col = j; col < m; col++) {
                (*h_Q)[row * m + col] = (*h_Q)[row * m + col] - 2.0 * h_A[(row + 1) * n + j] * temp[col - j];
            }
        }
    }
}

void h_q_panel_backward_accumulation(float* h_A, float** panel_Q, int m, int n, int global_offset, int panel_width) {
    /*
    * "Backward accumulation" of Q from householder vectors stored in lower trapezoidal region
    *   of A, after householder QR
    *
    * Reference:
    *   Golub, Van Loan. Matrix Computations, Fourth Edition. The Johns Hopkins
    *   University Press. Pg. 238. Algorithm 5.1.5
    */

    // Initialize panel Q as identity (always a square matrix)
    int panel_q_dim = m - global_offset;
    * panel_Q = (float*)malloc(panel_q_dim * panel_q_dim * sizeof(float));
    h_identity_mtx(*panel_Q, panel_q_dim, panel_q_dim);

    // Declare temporary vectors
    float* v;
    float beta;

    for (int j = global_offset + panel_width - 1; j >= global_offset; j--) { // iterate over householder vectors stored in lower part of A
        int v_length = m - j; // v is the householder vector, smallest first
        v = (float*)malloc(v_length * sizeof(float));

        // Q = (Im - 2v(v^T))Q
        // Q = Q_j:m,j:m - 2V @ ((V^T) @ Q_j:m,j:m)

        // (V^T) @ Q_j:m,j:m
        float* temp = (float*)malloc(v_length * sizeof(float));
        for (int col = j; col < m; col++) { // col of matrix A, where v is stored
            float inner_product = 0;
            for (int row = j; row < m; row++) { // rows of matrix A, where v is stored
                inner_product += 
                    h_A[(row + 1) * n + j] * (*panel_Q)[(row-global_offset) * panel_q_dim + (col-global_offset)];
            }
            temp[col - j] = inner_product;
        }

        // Q_j:m,j:m = Q_j:m,j:m - 2 * V @ ((V^T) @ Q_j:m,j:m)
        for (int row = j; row < m; row++) { // row of Q result
            for (int col = j; col < m; col++) { // col of Q result
                (*panel_Q)[(row-global_offset) * panel_q_dim + (col-global_offset)] = 
                    (*panel_Q)[(row-global_offset) * panel_q_dim + (col-global_offset)] - 
                            2.0 * h_A[(row + 1) * n + j] * temp[col - j];
            }
        }
    }
}

void h_wy_transform(float* h_A, float** h_Q, int m, int n, int global_offset, int panel_width)
{
    nvtxRangePush(__func__);
    float* W = (float*)malloc((m - global_offset) * panel_width * sizeof(float));
    float* Y = (float*)malloc((m - global_offset) * panel_width * sizeof(float));
    float* z = (float*)malloc((m - global_offset) * sizeof(float));
    float* W_Yt = (float*)malloc((m - global_offset) * (m - global_offset) * sizeof(float));

    // Dimensions of final result Im - WY^T, square
    int W_Yt_dim = m - global_offset;

    // Y = w_1
    for (int i = 0; i < W_Yt_dim; i++) {
        Y[i * panel_width] = h_A[(i + global_offset + 1) * n + global_offset];
        W[i * panel_width] = 2 * h_A[(i + global_offset + 1) * n + global_offset];
    }

    clock_t cycles = clock();

    // Iterate over columns of panel and update W, Y
    for (int i = 1; i < panel_width; i++) { // cols of panel
        // Calculate z = 2 * (I_m - WY^T)w_i
        // Im - WY^T (classic "triply-nested-loop")
        // Flops: (m-global_offset)x(m-global_offset)x(i)
        for (int row = 0; row < W_Yt_dim; row++) { // rows of W_Yt
            int row_offset = row * panel_width;
            for (int col = i; col < W_Yt_dim; col++) { // cols of W_Yt
                int col_offset = col * panel_width;
                // compute each inner product
                float inner_product = 0;
                for (int idx = 0; idx < i; idx++) { // idx of columns of W
                    inner_product += W[row_offset + idx] * Y[col_offset + idx];
                }
                if (row == col) { // Im is 1
                    W_Yt[row * W_Yt_dim + col] = 1 - inner_product; // Im - WY^T
                }
                else { // Im is zero
                    W_Yt[row * W_Yt_dim + col] = -inner_product;
                }
            }
        }

        // 2 * (Im - WY^T)w_i (matrix-vector product)
        // Flops: (m-global_offset)x(m-global_offset-i)
        for (int row = 0; row < W_Yt_dim; row++) {
            float inner_product = 0;
            for (int col = i; col < W_Yt_dim; col++) {
                inner_product += W_Yt[row * W_Yt_dim + col] * h_A[(global_offset + col + 1) * n + global_offset + i];
            }
            z[row] = 2 * inner_product;
        }

        // Copy z to W
        // Flops: (m-global_offset)
        for (int idx = 0; idx < W_Yt_dim; idx++) {
            if (idx < (i)) {
                Y[idx * panel_width + i] = 0;
            }
            else {
                Y[idx * panel_width + i] = h_A[(global_offset + idx + 1) * n + global_offset + i];
            }
            W[idx * panel_width + i] = z[idx];
        }
    }

    // Im - WY^T (classic "triply-nested-loop")
    // Flops: (m-global_offset)x(m-global_offset)xpanel_width
    for (int row = 0; row < W_Yt_dim; row++) { // rows of W_Yt
        for (int col = 0; col < W_Yt_dim; col++) { // cols of W_Yt
            // compute each inner product
            float inner_product = 0;
            for (int idx = 0; idx < panel_width; idx++) { // cols of W
                inner_product += W[row * panel_width + idx] * Y[col * panel_width + idx];
            }
            if (row == col) { // Im is 1
                W_Yt[row * W_Yt_dim + col] = 1 - inner_product; // Im - WY^T
            }
            else { // Im is zero
                W_Yt[row * W_Yt_dim + col] = -inner_product;
            }
        }
    }

    free(W);
    free(Y);
    free(z);
    //free(W_Yt);
    *h_Q = W_Yt;
    nvtxRangePop();
}

template <typename T>
__global__
void dev_cpy_strided_array(T* dest, T* src, int dest_height, int dest_width, int src_height, int src_width, int mode) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int smaller_width;
    int smaller_height;
    int larger_width;
    int larger_height;

    if (src_width < dest_width) {
        smaller_width = src_width;
        larger_width = dest_width;
    }
    else {
        smaller_width = dest_width;
        larger_width = src_width;
    }

    if (src_height < dest_height) {
        smaller_height = src_height;
        larger_height = dest_height;
    }
    else {
        smaller_height = dest_height;
        larger_height = src_height;
    }


    if (mode == TOP_LEFT) {
        if (row < smaller_height && col < smaller_width) {
            dest[row * dest_width + col] = src[row * src_width + col];
        }
        else if (row < dest_height && col < dest_width) {
            dest[row * dest_width + col] = (T)0.0;
        }
    }
    else if (mode == BOTTOM_RIGHT) {
        int row_offset = (larger_height - smaller_height);
        int col_offset = (larger_width - smaller_width);

        if (row >= row_offset && col >= col_offset && row < dest_height && col < dest_width) {
            dest[row * dest_width + col] = src[(row - row_offset) * src_width + col - col_offset];
        }
    }

}

template __global__ void dev_cpy_strided_array<float>(float*, float*, int, int, int, int, int);

template <typename T>
void h_launch_cpy_strided_array(T* h_dest, T* h_src, int dest_height, int dest_width, int src_height, int src_width) {

    // Allocate device memory
    T* dev_dest;
    T* dev_src;

    hipMalloc(&dev_dest, dest_width * dest_height * sizeof(T));
    hipMalloc(&dev_src, src_width * src_height * sizeof(T));

    hipMemcpy(dev_src, h_src, src_width * src_height * sizeof(T), hipMemcpyHostToDevice);

    // Configure grid of thread blocks
    int grid_height = dest_height / CPY_ARRAY_BLOCK_HEIGHT +
        (dest_height % CPY_ARRAY_BLOCK_HEIGHT != 0); // Integer div. rounded up
    int grid_width = dest_width / CPY_ARRAY_BLOCK_WIDTH +
        (dest_width % CPY_ARRAY_BLOCK_WIDTH != 0);

    dim3 gridDim(grid_height, grid_width, 1);
    dim3 blockDim(CPY_ARRAY_BLOCK_WIDTH, CPY_ARRAY_BLOCK_HEIGHT, 1);
    dev_cpy_strided_array<T> << <gridDim, blockDim >> > (dev_dest, dev_src, dest_height, dest_width, src_height, src_width, TOP_LEFT);

    hipDeviceSynchronize();

    hipMemcpy(h_dest, dev_dest, dest_height * dest_width * sizeof(T), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(dev_dest);
    hipFree(dev_src);
}

__global__ void global_mem_mmult(float* c_mtx, float* a_mtx, float* b_mtx, int a_width, int a_height, int b_width)
/*
* Computes result c matrix from the matrix multiplication C = AB using global memory with CUDA
*
* Assumed a_width == b_height
*
* Dimensions of C are a_height x b_width
*/
{
    // row and column of the C result
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < a_height && col < b_width) {
        // Calculate the inner product of the row of A and column of B
        float innerProduct = 0;
        for (int i = 0; i < a_width; i++) {
            innerProduct += a_mtx[a_width * row + i] * b_mtx[b_width * i + col];
        }

        c_mtx[b_width * row + col] = innerProduct;
    }
}

#define TILE_WIDTH 32

__global__ void shared_mem_mmult(float* c_mtx, float* a_mtx, float* b_mtx, int a_width, int a_height, int b_width)
/*
* Computes result c matrix from the matrix multiplication C = AB using shared memory with CUDA
*
* Assumed a_width == b_height
*
* Dimensions of C are a_height x b_width
*/
{
    // row and column of C result
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float bds[TILE_WIDTH][TILE_WIDTH];

    int ty = threadIdx.y, tx = threadIdx.x;

    int phases = ceil(a_width / (float)TILE_WIDTH);

    float pval = 0.0;
    for (int i = 0; i < phases; i++) {
        if ((i * TILE_WIDTH + tx < a_width) && (row < a_height)) {
            ads[ty][tx] = a_mtx[row * a_width + i * TILE_WIDTH + tx];
        }

        if ((i * TILE_WIDTH + ty < a_width) && (col < b_width)) {
            bds[ty][tx] = b_mtx[(i * TILE_WIDTH + ty) * b_width + col];
        }

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            if ((i * TILE_WIDTH + k) < a_width)
                pval += ads[ty][k] * bds[k][tx];
        }
        __syncthreads();
    }

    if (col < b_width && row < a_height) {
        c_mtx[row * b_width + col] = pval;
    }
}

__global__ 
void shared_mem_mmult_in_place(float* c_mtx, float* a_mtx, float* b_mtx, int m, int n, int k, int b_height, int b_width)
/*
* Computes result c matrix from the matrix multiplication C = AB using shared memory with CUDA
*
* Dimensions:
* A : m x k
* B : b_height x b_width => operate on bottom-right corner k x n submatrix
* C : m x n
*/
{
    // row and column of C result
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // offsets for 'in-place' read from b matrix
    int b_row_offset = b_height - k;
    int b_col_offset = b_width - n;

    __shared__ float ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float bds[TILE_WIDTH][TILE_WIDTH];

    int ty = threadIdx.y, tx = threadIdx.x;

    int phases = ceil(k / (float)TILE_WIDTH);

    float pval = 0.0;
    for (int i = 0; i < phases; i++) {
        if ((i * TILE_WIDTH + tx < k) && (row < m)) {
            ads[ty][tx] = a_mtx[row * k + i * TILE_WIDTH + tx];
        }

        if ((i * TILE_WIDTH + ty < k) && (col < n)) {
            bds[ty][tx] = b_mtx[(i * TILE_WIDTH + ty + b_row_offset) * b_width + (col + b_col_offset)];
        }

        __syncthreads();

        for (int idx = 0; idx < TILE_WIDTH; idx++) {
            if ((i * TILE_WIDTH + idx) < k)
                pval += ads[ty][idx] * bds[idx][tx];
        }
        __syncthreads();
    }

    if (col < n && row < m) {
        c_mtx[row * n + col] = pval;
    }

    __syncthreads();
}

__global__
void shared_mem_mmult_in_place_transpose_a(float* c_mtx, float* a_mtx, float* b_mtx, int m, int n, int k, int b_height, int b_width)
/*
* Computes result c matrix from the matrix multiplication C = AB using shared memory
* 
* assumed m = k
*
* Dimensions:
* A : m x k
* B : b_height x b_width => operate on bottom-right corner k x n submatrix``1`1~!`
* C : m x n
*/
{
    // row and column of C result
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // offsets for 'in-place' read from b matrix
    int b_row_offset = b_height - k;
    int b_col_offset = b_width - n;

    __shared__ float ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float bds[TILE_WIDTH][TILE_WIDTH];

    int ty = threadIdx.y, tx = threadIdx.x;

    int phases = ceil(k / (float)TILE_WIDTH);

    float pval = 0.0;
    for (int i = 0; i < phases; i++) {
        if ((i * TILE_WIDTH + tx < m) && (row < k)) {
            ads[tx][ty] = a_mtx[(i * TILE_WIDTH + tx) * k + row];
        }

        if ((i * TILE_WIDTH + ty < k) && (col < n)) {
            bds[ty][tx] = b_mtx[(i * TILE_WIDTH + ty + b_row_offset) * b_width + (col + b_col_offset)];
        }

        __syncthreads();

        for (int idx = 0; idx < TILE_WIDTH; idx++) {
            if ((i * TILE_WIDTH + idx) < m)
                pval += ads[idx][ty] * bds[idx][tx];
        }
        __syncthreads();
    }

    if (col < n && row < m) {
        c_mtx[row * n + col] = pval;
    }

    __syncthreads();
}

__global__
void dev_tensorcore_mmult_1_warp(float* c_mtx, half* a_mtx, half* b_mtx) {
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;

    using namespace nvcuda;
    // Create fragments 
    wmma::fragment<wmma::matrix_a, TC_TILE_M, TC_TILE_N, TC_TILE_K, half, wmma::row_major> Amat;
    wmma::fragment<wmma::matrix_b, TC_TILE_M, TC_TILE_N, TC_TILE_K, half, wmma::row_major> Bmat;
    wmma::fragment<wmma::accumulator, TC_TILE_M, TC_TILE_N, TC_TILE_K, float, void> Cmat;

    // Initialize output to zero
    wmma::fill_fragment(Cmat, 0.0f);

    // Load inputs
    wmma::load_matrix_sync(Amat, a_mtx, TC_TILE_M);
    wmma::load_matrix_sync(Bmat, b_mtx, TC_TILE_K);

    // Perfrom matrix multiplication
    wmma::mma_sync(Cmat, Amat, Bmat, Cmat);

    // Store output
    wmma::store_matrix_sync(c_mtx, Cmat, TC_TILE_N, wmma::mem_row_major);
}

template <typename T_A, typename T_B, typename T_C>
__global__
void dev_tensorcore_mmult_tiled(T_C* c_mtx, T_A* a_mtx, T_B* b_mtx, int m, int n, int k) {
    /*
    * Tiled matrix multiply using warp matrix multiply-accumulate (wmma)
    * 
    * The output matrix is divided into tiles (M-N-K), where each warp is responsible for computing one output tile
    */

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    using namespace nvcuda;

    // Determine warp index
    int warp_x = x / WARP_SIZE;
    int warp_y = y;

    int tiles_x = n / TC_TILE_N + (n % TC_TILE_N != 0);
    int tiles_y = m / TC_TILE_M + (m % TC_TILE_M != 0);
    int num_phases = k / TC_TILE_K + (k % TC_TILE_K != 0);

    if (warp_x < tiles_x && warp_y < tiles_y) {
        // Create fragments
        wmma::fragment<wmma::matrix_a, TC_TILE_M, TC_TILE_N, TC_TILE_K, T_A, wmma::row_major> Amat;
        wmma::fragment<wmma::matrix_b, TC_TILE_M, TC_TILE_N, TC_TILE_K, T_B, wmma::row_major> Bmat;
        wmma::fragment<wmma::accumulator, TC_TILE_M, TC_TILE_N, TC_TILE_K, T_C, void> Cmat;

        // Initialize output to zero
        wmma::fill_fragment(Cmat, 0.0f);

        // Compute tiled matrix multiply for warp
        for (int phase = 0; phase < num_phases; phase++) {
            // Load inputs
            T_A* a_idx = &a_mtx[warp_y * k * TC_TILE_M + phase * TC_TILE_K];
            T_B* b_idx = &b_mtx[phase * n * TC_TILE_K + warp_x * TC_TILE_N];

            wmma::load_matrix_sync(Amat, a_idx, k);
            wmma::load_matrix_sync(Bmat, b_idx, n);

            // Perfrom matrix multiplication, accumulate into C
            wmma::mma_sync(Cmat, Amat, Bmat, Cmat);
        }

        // Write output
        T_C* c_idx = &c_mtx[warp_y * n * TC_TILE_M + warp_x * TC_TILE_N];
        wmma::store_matrix_sync(c_idx, Cmat, n, wmma::mem_row_major);
    }
}


void test_tensorcore_mmult_gmem() {
    printf("\nTesting tensorcore 16x16x16 mmult...\n");

    __half* a_mtx = (__half*)malloc(16 * 16 * sizeof(__half));
    __half* b_mtx = (__half*)malloc(16 * 16 * sizeof(__half));
    float* c_mtx = (float*)malloc(16 * 16 * sizeof(float));

    // initialize matrices A, B, C
    for (int i = 0; i < 16; i++) {
        for (int j = 0; j < 16; j++) {
            a_mtx[i * 16 + j] = (__half)(float)j;
            b_mtx[i * 16 + j] = (__half)(float)j;
            c_mtx[i * 16 + j] = (__half)0.0f;
        }
    }

    // Allocate device memory
    __half* dev_a;
    __half* dev_b;
    float* dev_c;

    hipMalloc(&dev_a, 16 * 16 * sizeof(__half));
    hipMalloc(&dev_b, 16 * 16 * sizeof(__half));
    hipMalloc(&dev_c, 16 * 16 * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(dev_a, a_mtx, 16 * 16 * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_mtx, 16 * 16 * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c_mtx, 16 * 16 * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(32, 1, 1); // one warp

    dev_tensorcore_mmult_1_warp << <gridDim, blockDim >> > (dev_c, dev_b, dev_a);

    hipDeviceSynchronize();

    hipMemcpy(c_mtx, dev_c, 16 * 16 * sizeof(float), hipMemcpyDeviceToHost);

    // test result
    for (int i = 0; i < 16; i++) {
        for (int j = 0; j < 16; j++) {
            assert(c_mtx[i * 16 + j] == j * 120);
        }
    }

    printf("Test passed.\n");

}


void test_dev_smem_mmult(int m, int n, int k) {
    printf("\nTesting GPU SMEM tiled mmult %dx%dx%d...\n", m, n, k);

    float* a_mtx = (float*)malloc(m * k * sizeof(float));
    float* b_mtx = (float*)malloc(k * n * sizeof(float));
    float* c_mtx = (float*)malloc(m * n * sizeof(float));

    // initialize matrix A
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            a_mtx[i * k + j] = (float)(float)j;
        }
    }

    // initialize matrix B
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            b_mtx[i * n + j] = (float)(float)j;
        }
    }

    // initialize matrix C
    memset(c_mtx, 0.0, m * n * sizeof(float));

    // Allocate device memory
    float* dev_a;
    float* dev_b;
    float* dev_c;

    hipMalloc(&dev_a, m * k * sizeof(float));
    hipMalloc(&dev_b, k * n * sizeof(float));
    hipMalloc(&dev_c, m * n * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(dev_a, a_mtx, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_mtx, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c_mtx, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid
    dim3 gridDim((int)ceil((float)n / TILE_WIDTH), (int)ceil((float)m / TILE_WIDTH), 1);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1); // one warp

    shared_mem_mmult << <gridDim, blockDim >> > (dev_c, dev_a, dev_b, k, m, n);

    hipDeviceSynchronize();

    hipMemcpy(c_mtx, dev_c, m * n * sizeof(float), hipMemcpyDeviceToHost);

    float row_sum = 0;
    for (int i = 0; i < k; i++) {
        row_sum += i;
    }

    // test result
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            assert(abs(c_mtx[i * n + j] - (j * row_sum)) <= 1E-7 * row_sum * j * m);
        }
    }

    printf("Test passed.\n");
}

void test_dev_smem_mmult_in_place(int m, int n, int k, int b_width, int b_height) {
    /*
    * Computes C = A @ B', where b' (mxk) is stored in the "bottom-right" submatrix of a larger matrix B 
    * (b_height x b_width)
    */
    printf("\nTesting GPU SMEM tiled mmult (in-place) %dx%dx%d...\n", m, n, k);

    float* a_mtx = (float*)malloc(m * k * sizeof(float));
    float* b_mtx = (float*)malloc(b_height * b_width * sizeof(float));
    float* c_mtx = (float*)malloc(m * n * sizeof(float));

    // initialize matrix A
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            a_mtx[i * k + j] = (float)(float)j;
        }
    }

    memset(b_mtx, 0.0, b_width * b_height * sizeof(float));

    // initialize matrix B
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            b_mtx[(i + (b_height - k)) * b_width + (j + (b_width - n))] = (float)(float)j;
        }
    }

    // initialize matrix C
    memset(c_mtx, 0.0, m * n * sizeof(float));

    // Allocate device memory
    float* dev_a;
    float* dev_b;
    float* dev_c;

    hipMalloc(&dev_a, m * k * sizeof(float));
    hipMalloc(&dev_b, b_height * b_width * sizeof(float));
    hipMalloc(&dev_c, m * n * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(dev_a, a_mtx, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_mtx, b_height * b_width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c_mtx, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid
    dim3 gridDim((int)ceil((float)n / TILE_WIDTH), (int)ceil((float)m / TILE_WIDTH), 1);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1); // one warp

    shared_mem_mmult_in_place << <gridDim, blockDim >> > (dev_c, dev_a, dev_b, m, n, k, b_height, b_width);

    hipDeviceSynchronize();

    dim3 gridDim2((int)ceil((float)b_width / TILE_WIDTH), (int)ceil((float)b_height / TILE_WIDTH), 1);
    dim3 blockDim2(TILE_WIDTH, TILE_WIDTH, 1);
    dev_cpy_strided_array<float> << <gridDim2, blockDim2 >> >(dev_b, dev_c, b_height, b_width, m, n, BOTTOM_RIGHT);

    hipDeviceSynchronize();

    hipMemcpy(b_mtx, dev_b, b_height * b_width * sizeof(float), hipMemcpyDeviceToHost);

    float row_sum = 0;
    for (int i = 0; i < k; i++) {
        row_sum += i;
    }

    // test result
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            assert(abs(b_mtx[(i + (b_height - k)) * b_width + j + (b_width - n)] - (j * row_sum)) <= 1E-7 * row_sum * j * m);
        }
    }

    printf("Test passed.\n");
}

void test_dev_smem_mmult_in_place_transpose_a(int m, int n, int k, int b_width, int b_height) {
    /*
    * Computes C = A^T @ B', where b' (mxk) is stored in the "bottom-right" submatrix of a larger matrix B
    * (b_height x b_width), and A is transposed in memory
    */
    printf("\nTesting GPU SMEM tiled mmult (in-place & transposed A) %dx%dx%d...\n", m, n, k);

    float* a_mtx = (float*)malloc(m * k * sizeof(float));
    float* b_mtx = (float*)malloc(b_height * b_width * sizeof(float));
    float* c_mtx = (float*)malloc(m * n * sizeof(float));

    // initialize matrix A
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            a_mtx[i * k + j] = (float)(float)i;
        }
    }

    memset(b_mtx, 0.0, b_width * b_height * sizeof(float));

    // initialize matrix B
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            b_mtx[(i + (b_height - k)) * b_width + (j + (b_width - n))] = (float)(float)j;
        }
    }

    // initialize matrix C
    memset(c_mtx, 0.0, m * n * sizeof(float));

    // Allocate device memory
    float* dev_a;
    float* dev_b;
    float* dev_c;

    hipMalloc(&dev_a, m * k * sizeof(float));
    hipMalloc(&dev_b, b_height * b_width * sizeof(float));
    hipMalloc(&dev_c, m * n * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(dev_a, a_mtx, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_mtx, b_height * b_width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c_mtx, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid
    dim3 gridDim((int)ceil((float)n / TILE_WIDTH), (int)ceil((float)m / TILE_WIDTH), 1);
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1); // one warp

    shared_mem_mmult_in_place_transpose_a << <gridDim, blockDim >> > (dev_c, dev_a, dev_b, m, n, k, b_height, b_width);

    hipDeviceSynchronize();

    dim3 gridDim2((int)ceil((float)b_width / TILE_WIDTH), (int)ceil((float)b_height / TILE_WIDTH), 1);
    dim3 blockDim2(TILE_WIDTH, TILE_WIDTH, 1);
    dev_cpy_strided_array<float> << <gridDim2, blockDim2 >> > (dev_b, dev_c, b_height, b_width, m, n, BOTTOM_RIGHT);

    hipDeviceSynchronize();

    hipMemcpy(b_mtx, dev_b, b_height * b_width * sizeof(float), hipMemcpyDeviceToHost);

    float row_sum = 0;
    for (int i = 0; i < k; i++) {
        row_sum += i;
    }

    // test result
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            assert(abs(b_mtx[(i + (b_height - k)) * b_width + j + (b_width - n)] - (j * row_sum)) <= 1E-7 * row_sum * j * m);
        }
    }

    printf("Test passed.\n");
}

template <typename T_A, typename T_B, typename T_C>
void h_launch_dev_tensorcore_mmult_tiled(T_A* a_mtx, T_B* b_mtx, T_C* c_mtx, int m, int n, int k) {
    /*
    * Performs tiled matrix multiply C = A@B with TensorCore
    * 
    * Dimensions of A: mxk
    * Dimensions of B: kxn
    * Dimensions of C: mxn
    */

    // Allocation size must be integer multiple of TC tile size
    int m_padded = (m % TC_TILE_M) ? m + (TC_TILE_M - m % TC_TILE_M): m; // Padded height of A & C
    int n_padded = (n % TC_TILE_N) ? n + (TC_TILE_N - n % TC_TILE_N): n; // Padded width of B & C
    int k_padded = (k % TC_TILE_K) ? k + (TC_TILE_K - k % TC_TILE_K): k; // Padded inner dimension

    // Matrix sizes in bytes
    size_t a_bytes = m_padded * k_padded * sizeof(T_A);
    size_t b_bytes = k_padded * n_padded * sizeof(T_B);
    size_t c_bytes = m_padded * n_padded * sizeof(T_C);

    // Allocate host-side padded arrays
    T_A* h_a = (T_A*)malloc(a_bytes);
    T_B* h_b = (T_B*)malloc(b_bytes);
    T_C* h_c = (T_C*)malloc(c_bytes);

    // Pad arrays
    h_launch_cpy_strided_array<T_A>(h_a, a_mtx, m_padded, k_padded, m, k);
    h_launch_cpy_strided_array<T_B>(h_b, b_mtx, k_padded, n_padded, k, n);
    // Set output to zeros
    memset(h_c, 0, c_bytes);

    // Allocate input & output matrices on device
    T_A* dev_a;
    T_B* dev_b;
    T_C* dev_c;

    hipMalloc(&dev_a, a_bytes);
    hipMalloc(&dev_b, b_bytes);
    hipMalloc(&dev_c, c_bytes);

    // Copy matrices from host to device
    hipMemcpy(dev_a, h_a, a_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, h_b, b_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, h_c, c_bytes, hipMemcpyHostToDevice);

    // Configure grid of "warp blocks" which overlay output C
    int warp_grid_height = m / TC_TILE_M + (m % TC_TILE_M != 0);
    int warp_grid_width = n / TC_TILE_N + (n % TC_TILE_N != 0);

    // Configure grid of thread blocks
    int grid_height = warp_grid_height / TC_MMULT_THREAD_BLOCK_HEIGHT + 
                      (warp_grid_height % TC_MMULT_THREAD_BLOCK_HEIGHT != 0); // Integer div. rounded up
    int grid_width = warp_grid_width * WARP_SIZE / TC_MMULT_THREAD_BLOCK_WIDTH + 
                     (warp_grid_width % TC_MMULT_THREAD_BLOCK_WIDTH != 0);

    // Configure grid
    dim3 gridDim(grid_height, grid_width, 1);
    dim3 blockDim(TC_MMULT_THREAD_BLOCK_WIDTH, TC_MMULT_THREAD_BLOCK_HEIGHT, 1);

    dev_tensorcore_mmult_tiled<T_A, T_B, T_C> << <gridDim, blockDim >> > (dev_c, dev_b, dev_a, m_padded, n_padded, k_padded);

    hipDeviceSynchronize();

    hipMemcpy(h_c, dev_c, c_bytes, hipMemcpyDeviceToHost);

    h_launch_cpy_strided_array<T_C>(c_mtx, h_c, m, n, m_padded, n_padded);
}

template void h_launch_dev_tensorcore_mmult_tiled<half, half, float>(half*, half*, float*, int, int, int);

void test_template_tensorcore_mmult_tiled() {
    printf("\nTesting template tensorcore tiled mmult 33x33x33...\n");

    
    int m = 33;
    int n = 33;
    int k = 33;

    __half* a_mtx = (__half*)malloc(m * k * sizeof(__half));
    __half* b_mtx = (__half*)malloc(k * n * sizeof(__half));
    float* c_mtx = (float*)malloc(m * n * sizeof(float));

    // initialize matrices A, B, C
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            a_mtx[i * n + j] = (__half)(float)j;
            b_mtx[i * n + j] = (__half)(float)j;
            c_mtx[i * n + j] = (float)0.0f;
        }
    }

    h_launch_dev_tensorcore_mmult_tiled<half, half, float>(a_mtx, b_mtx, c_mtx, m, n, n);

    // test result
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            assert(c_mtx[i * n + j] == j * 528);
        }
    }

    printf("Test passed.\n");
}

void test_tensorcore_mmult_tiled() {
    printf("\nTesting tensorcore tiled mmult 32x32x32...\n");

    __half* a_mtx = (__half*)malloc(32 * 32 * sizeof(__half));
    __half* b_mtx = (__half*)malloc(32 * 32 * sizeof(__half));
    float* c_mtx = (float*)malloc(32 * 32 * sizeof(float));

    // initialize matrices A, B, C
    for (int i = 0; i < 32; i++) {
        for (int j = 0; j < 32; j++) {
            a_mtx[i * 32 + j] = (__half)(float)j;
            b_mtx[i * 32 + j] = (__half)(float)j;
            c_mtx[i * 32 + j] = (__half)0.0f;
        }
    }

    // Allocate device memory
    __half* dev_a;
    __half* dev_b;
    float* dev_c;

    hipMalloc(&dev_a, 32 * 32 * sizeof(__half));
    hipMalloc(&dev_b, 32 * 32 * sizeof(__half));
    hipMalloc(&dev_c, 32 * 32 * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(dev_a, a_mtx, 32 * 32 * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b_mtx, 32 * 32 * sizeof(__half), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c_mtx, 32 * 32 * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(64, 2, 1); // one warp

    dev_tensorcore_mmult_tiled << <gridDim, blockDim >> > (dev_c, dev_b, dev_a, 32, 32, 32);

    hipDeviceSynchronize();

    hipMemcpy(c_mtx, dev_c, 32 * 32 * sizeof(float), hipMemcpyDeviceToHost);

    // test result
    for (int i = 0; i < 32; i++) {
        for (int j = 0; j < 32; j++) {
            assert(c_mtx[i * 32 + j] == j * 496);
        }
    }

    printf("Test passed.\n");

}


__global__ 
void dev_householder_qr(float *dev_A, int m, int n, int global_offset) {
    /*
    * Computes the QR decomposition of A using Householder reflectors.
    * 
    * Reference: 
    *   Golub, Van Loan. Matrix Computations, Fourth Edition. The Johns Hopkins 
    *   University Press. Pg. 249. Algorithm 5.2.1
    */

    // Iterate over columns
    for (int k = global_offset; k < n; k++) {
        /*
        * Compute householder vector
        */

        // Skip last transform is square matrix
        if (m == n && k == n - 1) {
            break;
        }

        // Copy the column as u - can be done in parallel
        int len = m - k;
        float* u = (float*)malloc((len) * sizeof(float));
        for (int i = 0; i < len; i++) {
            u[i] = dev_A[n * (i+k) + k];
        }

        // Create the householder vector from the column vector
        int sign = 0;
        if (u[0] >= 0) {
            sign = 1;
        }
        else if (u[0] < 0) {
            sign = -1;
        }

        // Get the magnitude of u
        float mag = 0;
        for (int i = 0; i < len; i++) {
            mag+=u[i] * u[i];
        }
        mag = sqrtf(mag);

        // Compute householder normal vector w_k
        u[0] = sign * mag + u[0]; // v overwrites u
        // Normalize
        mag = 0;
        for (int i = 0; i < len; i++) { // TASK4 1 shashank: implement parallel algorithm in CUDA to replace for loop
            mag += u[i] * u[i];
        }
        mag = sqrtf(mag);
        for (int i = 0; i < len; i++) { // TASK5 1 shashank: implement parallel algorithm in CUDA to replace for loop
            u[i] /= mag; // w_k overwrites v, here u = w_k = v_k = householder vector
        }

        /*
        * Update trailing matrix : A_k:m, k : n = A_k:m,k:n - 2V((V ^ T)(A_k:m, k : n)
        */

        // (V^T)(A_k:m,k:n) - vector matrix product
        float* temp = (float*)malloc((n - k) * sizeof(float));
        for (int col = k; col < n; col++) { // TASK6 1 shashank: implement parallel algorithm in CUDA to replace for loop
            float inner_product = 0;
            for (int row = k; row < m; row++) {
                inner_product += u[row - k] * dev_A[row * n + col];
            }
            temp[col-k] = inner_product;
        }
        
        // (A_k:m,k:n) - 2 * (V)(V^T)(A_k:m,k:n)
        float* temp2 = (float*)malloc((n - k) * (m - k) * sizeof(float));
        for (int row = k; row < m; row++) { // TASK7 1 shashank: implement parallel algorithm in CUDA to replace for loop
            for (int col = k; col < n; col++) {
                temp2[(row - k) * (n - k) + (col - k)] = u[row-k] * temp[col-k];
                dev_A[row * n + col] = dev_A[row * n + col] - 2 * temp2[(row - k) * (n - k) + (col - k)];
            }
        }

        // Copy householder vector (vk) to lower triangular portion of A
        for (int row = k + 1; row < k + len + 1; row++) { // TASK8 1 shashank: implement parallel algorithm in CUDA to replace for loop
            dev_A[row * n + k] = u[row - k - 1];
        }

        free(temp);
        free(temp2);
        free(u);
    }
}

float* h_generate_random_matrix(int height, int width) {
    /*
    * Returns pointer to random float matrix of dimensions HeightxWidth
    */
    unsigned seed = time(0);
    srand(seed);
    float* matrix = (float*)malloc(height * width * sizeof(float));
    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            matrix[row * width + col] = rand(); // randomize this number
        }
    }

    return matrix;
}

void read_euroc_jacobian(const char filename[], int* rows, int* cols, double** matrix) {
    /*
    * Reads text file containing jacobian matrices from the Euroc dataset, and returns pointer to matrix
    */

    std::ifstream fin;

    std::string line;

    fin.open(filename);

    if (!fin) {
        printf("File not found.");
    }

    assert(fin);

    // Read first line to get dimensions
    getline(fin, line);

    std::cout << line << std::endl;
    int start = line.find(" ");
    int end = line.find(" ");

    std::string rows_str = line.substr(0, start);
    std::string cols_str = line.substr(start + 1, end);

    std::cout << rows_str << std::endl;
    std::cout << cols_str << std::endl;

    *cols = std::stoi(cols_str);
    *rows = std::stoi(rows_str);

    printf("Rows: %d\nCols: %d\n", *rows, *cols);

    // Allocate memory for matrix
    *matrix = (double*)malloc((*rows) * (*cols) * sizeof(double));

    for (int row = 0; row < (*rows); row++) {
        for (int col = 0; col < (*cols); col++) {
            (*matrix)[row * (*cols) + col] = (double)0.0;
        }
    }

    int linecount = 0;
    while (getline(fin, line)) {
        //std::cout << line << std::endl;

        std::wstring::size_type pos = line.find_first_not_of(' ');
        line = line.substr(pos);
        pos = line.find(' ');
        std::string row_idx_str = line.substr(0, pos);
        line = line.substr(pos);

        pos = line.find_first_not_of(' ');
        line = line.substr(pos);
        pos = line.find(' ');
        std::string col_idx_str = line.substr(0, pos);
        line = line.substr(pos);

        pos = line.find_first_not_of(' ');
        line = line.substr(pos);
        pos = line.find(' ');
        std::string val_str = line.substr(0, pos);

        //std::cout << row_idx_str << std::endl;
        //std::cout << col_idx_str << std::endl;
        //std::cout << val_str << std::endl;

        //printf("Row idx: %d\nCol idx: %d\nVal: %.15f\n", std::stoi(row_idx_str), std::stoi(col_idx_str), std::stod(val_str));

        int row_idx = std::stoi(row_idx_str);
        int col_idx = std::stoi(col_idx_str);
        double val = std::stod(val_str);

        (*matrix)[row_idx * (*cols) + col_idx] = val;
        linecount++;
    }

    printf("Total linecount: %d\n", linecount);
}




__global__
void dev_apply_qt_to_a(float* dev_A, float* dev_panel_Q, float* res_A, int m, int n, int tau, int lambda) {
    // Collaboratively load householder vectors vk from global memory to shared memory
    // Construct W, K from householder vectors
    // Construct Q
    // Collaboratively load matrix A to shared memory
    // Perform tiled GMMULT TensorCore warp-level mixed precision fused multiply add operations to update Q and A
    // Update matrix Q, A in global memory

    __shared__ float a_smem_tile[GMEM_MMULT_TILE_WIDTH][GMEM_MMULT_TILE_WIDTH];
    __shared__ float panel_q_smem_tile[GMEM_MMULT_TILE_WIDTH][GMEM_MMULT_TILE_WIDTH];

    // Row and column of the output result (A)
    int row = blockIdx.y * blockDim.y + threadIdx.y + lambda;
    int col = blockIdx.x * blockDim.x + threadIdx.x + tau;

    int panel_q_dim = (m - lambda); // panel_q is square matrix, shrinks for subsequent panels

    // Number of phases determined from block width
    int phases = panel_q_dim % GMEM_MMULT_TILE_WIDTH == 0 ? 
        panel_q_dim / GMEM_MMULT_TILE_WIDTH : panel_q_dim / GMEM_MMULT_TILE_WIDTH + 1;
    
    // Traverse phases and perform matrix-multiply accumulate into inner_product for each thread

    // check thread maps to output matrix
    bool valid_row = (row >= lambda && row < m); 
    bool valid_col = (col >= tau && col < n);

    // panel_Q[(inner_dim) * (m - lambda) + (row - lambda)] * A_old[(inner_dim + lambda) * n + col];

    float inner_product = 0;
    for (int p = 0; p < phases; p++) {
        // Check index doesn't exceed input bounds
        int a_idx_x = col;
        int a_idx_y = p * GMEM_MMULT_TILE_HEIGHT + row;
        int q_idx_x = (row - lambda);
        int q_idx_y = p * GMEM_MMULT_TILE_HEIGHT + (row-lambda);

        bool valid_idx_a = (a_idx_y < m);
        bool valid_idx_q = (q_idx_y < panel_q_dim);

        if (valid_idx_a && valid_idx_q && valid_row && valid_col) {
            // Collaboratively load data into smem
            a_smem_tile[threadIdx.y][threadIdx.x] = dev_A[a_idx_y * n + a_idx_x];
            panel_q_smem_tile[threadIdx.y][threadIdx.x] = dev_panel_Q[q_idx_y * panel_q_dim + q_idx_x];
        }

        __syncthreads();

        // Accumulate tile inner product
        if (valid_idx_a && valid_idx_q && valid_row && valid_col) {
            for (int i = 0; i < GMEM_MMULT_TILE_WIDTH; i++) {
                inner_product += panel_q_smem_tile[i][threadIdx.y] * a_smem_tile[i][threadIdx.x];
            }
        }

        __syncthreads();
    }

    
}

__global__ 
void dev_apply_qt_to_a_tensorcore_gmem(half* dev_A, half* dev_panel_Q, int m, int n, int tau, int lambda) {

}

__global__
void dev_apply_qpanel_to_q(float* dev_Q, float* dev_Q_panel, float* dev_Q_result, int m, int lambda) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x + lambda;

    if (row >= 0 && row < m && col >= lambda && col < m) {
        float inner_product = 0;
        for (int inner_dim = 0; inner_dim < (m - lambda); inner_dim++) {
            inner_product += dev_Q[row * m + inner_dim + lambda] * dev_Q_panel[(inner_dim * (m - lambda)) + (col - lambda)];
        }
        dev_Q_result[row * m + col] = inner_product;
    }
}

__global__
void dev_cpy_panel_result_a(float* dev_A, float* dev_A_panel_result, int m, int n, int tau, int lambda) {
    // Row and column of the output result (A)
    int row = blockIdx.y * blockDim.y + threadIdx.y + lambda;
    int col = blockIdx.x * blockDim.x + threadIdx.x + tau;

    int panel_a_height = (m - lambda); // panel_q is square matrix, shrinks for subsequent panels
    int panel_a_width = (n - tau);

    dev_A[row * n + col] = dev_A_panel_result[(row - lambda) * (panel_a_width) + (col - tau)];

    __syncthreads();
}


void dev_block_qr(float* A, float* Q, int m, int n, int r) {
    /*
    * GPU code to compute QR decomposition with Block QR algorithm
    */

    float* panel_Q = NULL;
    float* panel_Q_wy = NULL;
    int lambda = 0;
    while (lambda < n) { // panel starts at lambda
        int tau = (lambda + r < n) ? (lambda + r) : n; // panel ends at tau

        // Q is stored in factored form in lower triangular portion of dev_A
        // R is stored in upper triangular portion of dev_A
        h_householder_qr(A, m, n, lambda, tau-lambda);

        // Get panel Q from factors - dim panel_Q: (m-lambda)x(m-lambda)
        //h_wy_transform(A, &panel_Q_wy, m, n, lambda, tau-lambda); // TASK10 3 shashank: write cuda kernel to implement WY transform on GPU
        h_q_panel_backward_accumulation(A, &panel_Q, m, n, lambda, tau-lambda);

        // Update matrix A = Q^T @ A
        float blockWidth = 32.;
        float blockHeight = 32.;

        float* dev_A;
        float* dev_Q;
        float* dev_panel_Q;
        float* dev_A_panel_result;
        float* dev_Q_result;

        hipMalloc(&dev_A, m * n * sizeof(float));
        hipMalloc(&dev_Q, m * m * sizeof(float));
        hipMalloc(&dev_panel_Q, (m - lambda) * (m - lambda) * sizeof(float));
        hipMalloc(&dev_A_panel_result, (m - lambda) * (n - tau) * sizeof(float));
        hipMalloc(&dev_Q_result, m * m * sizeof(float));

        hipMemcpy(dev_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_panel_Q, panel_Q, (m - lambda) * (m - lambda) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_Q, Q, m * m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(dev_Q_result, Q, m * m * sizeof(float), hipMemcpyHostToDevice);

        dim3 BlockDim((int)blockWidth, (int)blockHeight, 1);
        dim3 GridDim(ceil((n - tau) / blockWidth), ceil((m - lambda) / blockHeight), 1);

        // Updates trailing matrix in place : A = Qt @ A
        shared_mem_mmult_in_place_transpose_a<<<GridDim, BlockDim>>>(dev_A_panel_result, dev_panel_Q, dev_A, 
                                                        (m - lambda), (n - tau), (m - lambda), m, n);

        hipDeviceSynchronize();

        dim3 gridDim2((int)ceil((float)n / TILE_WIDTH), (int)ceil((float)m / TILE_WIDTH), 1);
        dim3 blockDim2(TILE_WIDTH, TILE_WIDTH, 1);
        dev_cpy_strided_array<float> << <gridDim2, blockDim2 >> > (dev_A, dev_A_panel_result, m, n, 
                                                                  (m - lambda), (n - tau), BOTTOM_RIGHT);

        hipDeviceSynchronize();

        dim3 BlockDim3((int)blockWidth, (int)blockHeight, 1);
        dim3 GridDim3(ceil((m - lambda) / blockWidth), ceil((m) / blockHeight), 1);
        dev_apply_qpanel_to_q << <GridDim3, BlockDim3 >> >(dev_Q, dev_panel_Q, dev_Q_result, m, lambda);

        hipDeviceSynchronize();

        hipMemcpy(A, dev_A, m * n * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(Q, dev_Q_result, m * m * sizeof(float), hipMemcpyDeviceToHost);
        
        hipFree(dev_A);
        hipFree(dev_Q);
        hipFree(dev_panel_Q);
        hipFree(dev_A_panel_result);

        free(panel_Q);

        // increment panel offset
        lambda = tau;
    }
}


void test_dev_householder_qr(int m, int n, int r) {
    printf("\nTesting GPU householder QR...\n");
    printf("Dimensions of A: %dx%d\n", m, n);

    float* h_A_in = h_generate_random_matrix(m, n);

    float* h_A_out = (float*)malloc((m+1) * n * sizeof(float)); // extra row gives room for storing householder vectors in lower triangular portion of A
    float* h_Q_out = (float*)malloc(m * m * sizeof(float));
    float* h_R = (float*)malloc(m * n * sizeof(float));

    // Allocate device memory for input matrix
    float* dev_A;
    float* dev_Q; // Matrix Q in A=QR

    //hipMalloc(&dev_Q, m * m * sizeof(float));
    hipMalloc(&dev_A, (m+1) * n * sizeof(float));

    // Copy input matrix to device Global memory
    hipMemcpy(dev_A, h_A_in, m * n * sizeof(float), hipMemcpyHostToDevice);

    // Call kernel to collaboratively copy input matrix from Global memory to Shared memory
    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(1, 1, 1);
    // Time execution of the following kernel call
    clock_t cycles = clock(); // Time how long the QR function takes to execute
    dev_householder_qr <<<DimGrid, DimBlock >> > (dev_A, m, n, 0);
    hipDeviceSynchronize();
    cycles = clock() - cycles;
    float time_ms = cycles * 1000 / CLOCKS_PER_SEC;
    float flops = h_qr_flops_per_second(time_ms, m, n);

    hipMemcpy(h_A_out, dev_A, (m+1) * n * sizeof(float), hipMemcpyDeviceToHost);
    //hipMemcpy(h_Q_out, dev_Q, m * m * sizeof(float), hipMemcpyDeviceToHost);

    h_q_backward_accumulation(h_A_out, &h_Q_out, m, n);
    //h_wy_transform(h_A_out, &h_Q_out, m, n, 0, n);

    h_strip_R_from_A((float*)h_A_out, h_R, m, n);

    float backward_error = h_backward_error((float*)h_A_in, h_R, h_Q_out, m, n);
    float error3 = h_error_3(h_R, m, n);
    float error2 = h_error_2(h_Q_out, m);

    printf("GPU householder QR finished in %.2f ms...\n", time_ms);
}

void h_block_qr(float* A, float* Q, int m, int n, int r) {
    /*
    * Sequential version of the block QR algorithm, runs on CPU
    */

    // initialize Q, lambda, k
    //h_identity_mtx(Q, m, m);
    float* panel_Q = NULL;
    int lambda = 0;
    while (lambda < n) { // panel starts at lambda
        int tau = (lambda + r < n) ? (lambda + r) : n; // panel ends at tau

        // Q is stored in factored form in lower triangular portion of dev_A
        // R is stored in upper triangular portion of dev_A
        h_householder_qr(A, m, n, lambda, tau-lambda);

        // Get panel Q from factors
        //h_wy_transform(A, &panel_Q, m, n, lambda, tau-lambda); // dim panel_Q: (m-lambda)x(m-lambda)
        h_q_panel_backward_accumulation(A, &panel_Q, m, n, lambda, tau - lambda);

        // Update matrix A = Q^T @ A
        float* A_old = (float*)malloc(m * n * sizeof(float));
        memcpy(A_old, A, m * n * sizeof(float));
        for (int row = lambda; row < m; row++) {
            for (int col = tau; col < n; col++) {
                float inner_product = 0;
                for (int inner_dim = 0; inner_dim < (m - lambda); inner_dim++) {
                    inner_product += panel_Q[(inner_dim) * (m - lambda) + (row - lambda)] * A_old[(inner_dim + lambda) * n + col];
                }
                A[row * n + col] = inner_product;
            }
        }
        free(A_old);

        // Update global Q
        float* Q_old = (float*)malloc(m * m * sizeof(float)); 
        memcpy(Q_old, Q, m * m * sizeof(float));
        for (int row = 0; row < m; row++) {
            for (int col = lambda; col < m; col++) {
                float inner_product = 0;
                for (int inner_dim = 0; inner_dim < (m - lambda); inner_dim++) {
                    inner_product += Q_old[row * m + inner_dim + lambda] * panel_Q[(inner_dim * (m - lambda)) + (col - lambda)];
                }
                Q[row * m + col] = inner_product;
            }
        }
        free(Q_old);
        free(panel_Q);

        // increment panel offset
        lambda = tau;
    }
}



void test_h_mmult() {
    float A[3][3] = {
        {1, 2, 3},
        {1, 2, 3},
        {1, 2, 3}
    };

    int m = 3;
    int n = 3;
    int k = 3;

    float* C = (float*)malloc(m * n * sizeof(float));

    h_mmult((float*)A, (float*)A, C, m, n, k);
}

void test_h_mmult_transpose_A() {
    float A[3][3] = {
    {1, 2, 3},
    {1, 2, 3},
    {1, 2, 3}
    };

    float expected_result[3][3] = {
        {3, 6, 9},
        {6, 12, 18},
        {9, 18, 27}
    };

    int m = 3;
    int n = 3;
    int k = 3;

    float* C = (float*)malloc(m * n * sizeof(float));

    h_mmult_transpose_A((float*)A, (float*)A, C, m);

    for (int row = 0; row < m; row++) {
        for (int col = 0; col < n; col++) {
            assert((C[row * n + col] - ((float*)expected_result)[row * n + col]) < 1E-8);
        }
    }
}

void test_h_householder_qr(int m, int n, int r) {
    /*
    * Test host version of householder QR
    */

    // TASK14 3 alice: iterate over many matrix sizes, & test matrices from Tong
    printf("\nTesting sequential householder QR...\n");

    printf("Dimensions of A: %dx%d\n", m, n);

    float* A_in = h_generate_random_matrix(m, n);

    int global_offset = 0;

    float* Q = (float*)malloc(m * m * sizeof(float));
    float* R = (float*)malloc(m * n * sizeof(float));
    float* A_out = (float*)malloc((m + 1) * n * sizeof(float));

    h_matrix_cpy((float*)A_in, A_out, m, n);
    

    //h_block_qr((float*)A, Q, m, n, r);
    clock_t cycles = clock();
    h_householder_qr(A_out, m, n, 0, n);
    cycles = clock() - cycles;
    float time_ms = cycles * 1000 / CLOCKS_PER_SEC;
    float flops = h_qr_flops_per_second(time_ms, m, n);

    h_q_backward_accumulation(A_out, &Q, m, n);

    h_strip_R_from_A((float*)A_out, R, m, n);

    float backward_error = h_backward_error((float*)A_in, R, Q, m, n);
    float error3 = h_error_3(R, m, n);
    float error2 = h_error_2(Q, m);
    //printf("||A - QR||/||A|| = %e\n", backward_error);
    //printf("||QT @ Q - Im|| = %e\n", h_error_2(Q, m));
    //printf("||L|| = %e\n", error3);
    printf("Averaged %.2f GFLOPs\n", flops / 1E9);
    printf("Sequential householder finished in %.2f ms\n", time_ms);

    h_write_results_to_log(m, n, time_ms, flops, backward_error, "cpu_householder");


    // write results to log file
    free(Q);
    free(R);
    free(A_out);
    free(A_in);
}

/*
 * Test jacobian matrices for householder_qr
 * ||A - QR||/||A|| = -nan Error Criteria: False
 * uncertain reason for the error.
 */
void process_files_in_directory(const char *directory_path,
                                const char *file_prefix) {
  DIR *dir;
  struct dirent *ent;

  if ((dir = opendir(directory_path)) != NULL) {
    while ((ent = readdir(dir)) != NULL) {
      if (ent->d_type == DT_REG) {
        if (strncmp(ent->d_name, file_prefix, strlen(file_prefix)) == 0) {
          char path[512];
          snprintf(path, sizeof(path), "%s/%s", directory_path, ent->d_name);
          printf("Processing file: %s\n", path);

          int rows, cols;
          double *A_in;
          read_euroc_jacobian(path, &rows, &cols, &A_in);
          int m = rows;
          int n = cols;
          int global_offset = 0;

          float *Q = (float *)malloc(m * m * sizeof(float));
          float *R = (float *)malloc(m * n * sizeof(float));
          float *A_out = (float *)malloc((m + 1) * n * sizeof(float));

          h_matrix_cpy((float *)A_in, A_out, m, n);

          // Start timing
          auto start_time = std::chrono::high_resolution_clock::now();
          h_householder_qr((float *)A_out, m, n, 0, n);
          // Calculate elapsed time
          auto end_time = std::chrono::high_resolution_clock::now();
          std::chrono::duration<float, std::milli> elapsed_time =
              end_time - start_time;
          float time_ms = elapsed_time.count();
          float flops = h_qr_flops_per_second(time_ms, m, n);
          // Print elapsed time with appropriate unit
          std::string unit;
          float time_value;
          if (time_ms < 1000.0f) {
            time_value = time_ms;
            unit = "ms";
          } else if (time_ms < 60000.0f) {
            time_value = time_ms / 1000.0f;
            unit = "s";
          } else {
            time_value = time_ms / 60000.0f;
            unit = "min";
          }

          // h_wy_transform(A_out, &Q, m, n, global_offset, r);
          h_q_backward_accumulation(A_out, &Q, m, n);
          h_strip_R_from_A((float *)A_out, R, m, n);

          float backward_error = h_backward_error((float *)A_in, R, Q, m, n);
          float error3 = h_error_3(R, m, n);
          float error2 = h_error_2(Q, m);

          // printf("||A - QR||/||A|| = %e\n", backward_error);
          // printf("||QT @ Q - Im|| = %e\n", h_error_2(Q, m));
          // printf("||L|| = %e\n", error3);
          printf("Averaged %.4f GFLOPs\n", flops / 1E9);
          std::cout << "Sequential householder finished in " << std::fixed
                    << std::setprecision(4) << time_value << " " << unit
                    << std::endl;
          h_write_results_to_log(m, n, 0, 0, backward_error, "cpu_householder");

          free(Q);
          free(R);
          free(A_out);
          free(A_in);
        }
      }
    }
    closedir(dir);
  } else {
    perror("Unable to open directory");
  }
}

void test_h_jhouseholder_qr() {
  printf("\nTesting (jacobians) sequential householder QR...\n");
  const char *directory_path =
      "/home/qr/Desktop/MixedPrecisionBlockQR/Cuda/jacobians/";

  printf("Processing A_reduced files:\n");
  process_files_in_directory(directory_path, "A_reduced");

  // The execution is taking an amount of time
  // printf("\nProcessing A_0 files:\n");
  // process_files_in_directory(directory_path, "A_0");

  printf("\n(jacobians) Sequential householder QR finished...\n");
}



void test_h_wy_transform(int m, int n, int r) {
    // Initialize test matrix A input on Host
    // TASK16 Alice: iterate over many matrix sizes
    printf("Dimensions of A: %dx%d\n", m, n);

    float *h_A_in = h_generate_random_matrix(m, n);

    float* h_A_out = (float*)malloc((m + 1) * n * sizeof(float)); // extra row (m+1) gives room for storing householder vectors in lower triangular portion of A
    float* h_R = (float*)malloc(m * n * sizeof(float));
    float* h_Q_out = NULL; // pointer to Q is returned by h_wy_transform

    h_householder_qr((float*)h_A_in, m, n, 0, n);

    h_wy_transform((float*)h_A_out, &h_Q_out, m, n, 0, n);

    h_strip_R_from_A(h_A_out, h_R, m, n);

    float backward_error = h_backward_error((float*)h_A_in, h_R, h_Q_out, m, n);

    free(h_A_out);
    free(h_Q_out);
    free(h_R);
}


void test_h_block_qr(int m, int n, int r) {
    /*
    * Test host version of block QR
    */

    printf("\nTesting sequential block QR...\n");
    printf("Dimensions of A (m, n, r): (%d,%d,%d)\n", m, n, r);

    float* A_in = h_generate_random_matrix(m, n);

    float* Q = (float*)malloc(m * m * sizeof(float));
    float* R = (float*)malloc(m * n * sizeof(float));
    float* A_out = (float*)malloc((m + 1) * n * sizeof(float));

    h_identity_mtx(Q, m, m);

    h_matrix_cpy((float*)A_in, A_out, m, n);

    clock_t cycles = clock(); // Time how long the QR function takes to execute
    h_block_qr((float*)A_out, Q, m, n, r);
    cycles = clock() - cycles;
    float time_ms = cycles * 1000 / CLOCKS_PER_SEC;

    float flops_per_second = h_qr_flops_per_second(time_ms, m, n);

    h_strip_R_from_A((float*)A_out, R, m, n);

    float backward_error = h_backward_error((float*)A_in, R, Q, m, n);
    float error2 = h_error_2(Q, m);
    float error3 = h_error_3(R, m, n);

    // write results to log file
    h_write_results_to_log(m, n, time_ms, flops_per_second, backward_error, "cpu_block");

    printf("Sequential block QR finished in %.2f ms...\n", time_ms);
    //printf("||A - QR||/||A|| = %e\n", backward_error);
    free(Q);
    free(R);
    free(A_out);
}

struct QRProblemSize {
    // A = QR problem set dimensions
    int m; // height of matrix A
    int n; // width of matrix A
    int r; // block QR panel width
};

# define NUM_STATIC_QR_TESTS 21
# define NUM_STATIC_MMULT_TESTS 15

void test_qr(QR_FUNC f) {

    QRProblemSize testDim[NUM_STATIC_QR_TESTS] = {
        {6, 4, 2},
        {6, 4, 1},
        {6, 4, 3},
        {12, 8, 4},
        {12, 8, 5},
        {12, 8, 6},
        {12, 8, 2},
        {12, 8, 8},
        {12, 8, 3},
        {24, 16, 8},
        {24, 16, 12},
        {60, 40, 8},
        {60, 40, 16},
        {80, 80, 16},
        {97, 90, 16},
        {100, 80, 16},
        {128, 80, 16},
        {129, 80, 16},
        {240, 160, 16},
        {600, 400, 16},
        // {1800, 1800, 32}
    };

    for (int i = 0; i < NUM_STATIC_QR_TESTS; i++) {
        f(testDim[i].m, testDim[i].n, testDim[i].r);
    }
}

struct MMULTProblemSize {
    // C = A @ B problem set dimensions
    // Dimensions of A: m x k
    // Dimensions of B: k x n
    // Dimensions of C: m x n
    int m;
    int n;
    int k;
};

void test_mmult(MMULT_FUNC f) {
    QRProblemSize testDim[NUM_STATIC_MMULT_TESTS] = {
        {6, 4, 2},
        {6, 4, 1},
        {6, 4, 3},
        {12, 8, 4},
        {12, 8, 5},
        {12, 8, 6},
        {12, 8, 2},
        {12, 8, 8},
        {12, 8, 3},
        {24, 16, 8},
        {24, 16, 12},
        {60, 40, 8},
        {240, 160, 16},
        {600, 400, 16},
        {600, 400, 600}
    };

    for (int i = 0; i < NUM_STATIC_MMULT_TESTS; i++) {
        f(testDim[i].m, testDim[i].n, testDim[i].r);
    }
}

void test_mmult_in_place() {
    MMULTProblemSize testDim[7] = {
        {6, 4, 6},
        {12, 8, 12},
        {24, 16, 24},
        {60, 40, 60},
        {240, 160, 240},
        {400, 300, 400},
        {300, 300, 300}
    };

    for (int i = 0; i < 7; i++) {
        test_dev_smem_mmult_in_place(testDim[i].m, testDim[i].n, testDim[i].k, 400, 400);
    }
}

void test_mmult_in_place_transpose_a() {
    MMULTProblemSize testDim[7] = {
    {6, 4, 6},
    {12, 8, 12},
    {24, 16, 24},
    {60, 40, 60},
    {240, 160, 240},
    {400, 300, 400},
    {300, 300, 300}
    };

    for (int i = 0; i < 7; i++) {
        test_dev_smem_mmult_in_place_transpose_a(testDim[i].m, testDim[i].n, testDim[i].k, 400, 400);
    }
}

void test_dev_block_qr(int m, int n, int r) {
    /*
    * Test GPU version of block QR
    */

    printf("\nTesting GPU block QR...\n");
    printf("Dimensions of A (m, n, r): (%d, %d, %d)\n", m, n, r);

    float* A_in = h_generate_random_matrix(m, n);

    float* Q1 = (float*)malloc(m * m * sizeof(float));
    float* Q2 = (float*)malloc(m * m * sizeof(float));
    float* R = (float*)malloc(m * n * sizeof(float));
    float* A_out = (float*)malloc((m + 1) * n * sizeof(float));
    float* A_out2 = (float*)malloc((m + 1) * n * sizeof(float));

    h_identity_mtx(Q1, m, m);
    h_identity_mtx(Q2, m, m);

    h_matrix_cpy((float*)A_in, A_out, m, n);
    h_matrix_cpy((float*)A_in, A_out2, m, n);

    clock_t cycles = clock(); // Time how long the QR function takes to execute
    //h_block_qr((float*)A_out, Q1, m, n, r);
    dev_block_qr((float*)A_out2, Q2, m, n, r);
    cycles = clock() - cycles;

    float time_ms = cycles * 1000 / CLOCKS_PER_SEC;

    float flops = h_qr_flops_per_second(time_ms, m, n);

    h_strip_R_from_A((float*)A_out2, R, m, n);

    float backward_error = h_backward_error((float*)A_in, R, Q2, m, n);
    float error2 = h_error_2(Q2, m);
    float error3 = h_error_3(R, m, n);

    // write results to log file
    h_write_results_to_log(m, n, time_ms, flops, backward_error, "gpu_block");

    printf("GPU block QR finished...\n");
    printf("Averaged %.2f GFLOPs\n", flops / 1E9);
    printf("GPU Block QR finished in %.2f ms...\n", time_ms);
   // printf("||A - QR||/||A|| = %e\n", backward_error);
    
    free(Q1);
    free(Q2);
    free(R);
    free(A_out);
    free(A_out2);
    free(A_in);
}



int main() {
    test_h_mmult();
    test_h_mmult_transpose_A();
    test_h_jhouseholder_qr();

    test_qr(test_h_householder_qr);
    test_qr(test_dev_householder_qr);
    test_qr(test_h_block_qr);
    //test_qr(test_dev_block_qr);

    //test_mmult(test_dev_smem_mmult);
    //test_mmult_in_place();
    //test_mmult_in_place_transpose_a();
    test_qr(test_dev_block_qr);
    //test_mmult(test_dev_smem_mmult_in_place);

    //test_dev_smem_mmult(6000, 4000, 6000);
    //test_tensorcore_mmult_gmem();
    //test_tensorcore_mmult_tiled();
    //test_template_tensorcore_mmult_tiled();
    //test_dev_block_qr_tensorcore_gmem();
}
